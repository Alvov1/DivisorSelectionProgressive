#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <filesystem>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include "AesiMultiprecision.h"
#include "Timer.h"

std::vector<uint64_t> loadPrimes(const std::filesystem::path& fromLocation) {
    if(!std::filesystem::exists(fromLocation) || !std::filesystem::is_regular_file(fromLocation))
        throw std::invalid_argument("Failed to load prime table: bad input file");

    std::ifstream input(fromLocation, std::ios::binary);
    uint64_t buffer {}; input.read(reinterpret_cast<char*>(&buffer), sizeof(uint64_t));

    std::vector<uint64_t> primes (buffer);
    for(auto& prime: primes)
        input.read(reinterpret_cast<char*>(&prime), sizeof(uint64_t));

    return primes;
}

__global__ void kernel(const Aesi<512>* numberAndFactor, const uint64_t* const primes, std::size_t primesCount) {
    const auto threadId = blockDim.x * blockIdx.x + threadIdx.x,
            threads = gridDim.x * blockDim.x,
            max_it = 400000 / threads,
            bStart = 2 + blockIdx.x,
            bInc = gridDim.x,
            B_MAX = 2000000000U;

    const Aesi n = numberAndFactor[0], factor = numberAndFactor[1];
    const auto checkWriteRepeat = [&n](const Aesi<512> &value) {
        if (value < 2 || value >= n) return false;
        char buffer[512]{};
        value.getString<10>(buffer, 512);
        return printf("Found divisor: %s\n", buffer) > 14;
    };
    if(!factor.isZero()) return;

    Aesi a = threadId * max_it + 2, e = 1;
    for (unsigned B = bStart; B < B_MAX; B += bInc) {
        auto primeUl = primes[0];

        for (unsigned pi = 0; primeUl < B; ++pi) {
            if(!factor.isZero()) return;
            const unsigned power = log(static_cast<double>(B)) / log(static_cast<double>(primeUl));
            e *= static_cast<uint64_t>(pow(static_cast<double>(primeUl), static_cast<double>(power)));
            primeUl = primes[pi + 1];
        }

        if (e == 1) continue;

        for (unsigned it = 0; it < max_it; ++it) {
            if(!factor.isZero()) return;
            if (checkWriteRepeat(Aesi<512>::gcd(a, n)))
                return;

            if (checkWriteRepeat(Aesi<512>::gcd(Aesi<512>::powm(a, e, n) - 1, n)))
                return;

            a += threads * max_it;
        }
    }
}

int main(int argc, const char* const* const argv) {
    if(argc < 4)
        return std::printf("Usage: %s factorize <number> <primes location>", argv[0]);

    thrust::device_vector<Aesi<512>> numberAndFactor = { { std::string_view(argv[2]) }, {} };
    Timer::init() << "Factorizing number " << std::hex << std::showbase << numberAndFactor[0] << '.' << Timer::endl;

    const thrust::device_vector<uint64_t> primes = loadPrimes(argv[3]);
    Timer::out << "Loaded prime table of " << primes.size() << " elements." << Timer::endl;

    kernel<<<32, 32>>>(
            thrust::raw_pointer_cast(numberAndFactor.data()),
            thrust::raw_pointer_cast(primes.data()),
            primes.size());

    const auto code = hipDeviceSynchronize();
    if (code != hipSuccess)
        return std::printf("Kernel launch failed: %s.\n", hipGetErrorString(code));
    Timer::out << "Kernel completed. Founded factor: " << std::hex << std::showbase << numberAndFactor[1] << '.' << Timer::endl;

    return 0;
}
