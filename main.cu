#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <filesystem>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include "AesiMultiprecision.h"
#include "Timer.h"

std::vector<uint64_t> loadPrimes(const std::filesystem::path& fromLocation) {
    if(!std::filesystem::exists(fromLocation) || !std::filesystem::is_regular_file(fromLocation))
        throw std::invalid_argument("Failed to load prime table: bad input file");

    std::ifstream input(fromLocation, std::ios::binary);
    uint64_t buffer {}; input.read(reinterpret_cast<char*>(&buffer), sizeof(uint64_t));

    std::vector<uint64_t> primes (buffer);
    for(auto& prime: primes)
        input.read(reinterpret_cast<char*>(&prime), sizeof(uint64_t));

    return primes;
}

__global__ void kernel(Aesi<512>* const numberAndFactor, const uint64_t* const primes, std::size_t primesCount) {
    const auto threadId = blockDim.x * blockIdx.x + threadIdx.x,
            threads = gridDim.x * blockDim.x,
            max_it = 400000 / threads,
            bStart = 2 + blockIdx.x,
            bInc = gridDim.x,
            B_MAX = 2000000000U;

    const Aesi<512> n = numberAndFactor[0]; Aesi<512>* const factor = numberAndFactor + 1;

    const auto checkFactor = [&n, &factor, &threadId] (const Aesi<512>& candidate) {
        if(candidate > 1 && candidate < n) {
            factor->atomicSet(candidate);
            char buffer[100]{};
            candidate.getString<10>(buffer, 100);
            printf("Thread %d: found factor %s.\n", threadId, buffer);
            return true;
        } else return false;
    };

    Aesi a = threadId * max_it + 2, e = 1;
    for (unsigned B = bStart; B < B_MAX; B += bInc) {
        auto primeUl = primes[0];

        for (unsigned pi = 0; primeUl < B; ++pi) {
            if(!factor->isZero()) return;
            const unsigned power = log(static_cast<double>(B)) / log(static_cast<double>(primeUl));
            e *= static_cast<uint64_t>(pow(static_cast<double>(primeUl), static_cast<double>(power)));
            primeUl = primes[pi + 1];
        }

        if (e == 1) continue;

        for (unsigned it = 0; it < max_it; ++it) {
            if(!factor->isZero())
                return;

            if(checkFactor(Aesi<512>::gcd(a, n)))
                return;

            if(checkFactor(Aesi<512>::gcd(Aesi<512>::powm(a, e, n) - 1, n)))
                return;

            a += threads * max_it;
        }

        printf("Thread 0: 5 (%u).\n", B);
    }

    if(threadId % 64 == 0)
        printf("Thread %u exited.\n", threadId);
}

int main(int argc, const char* const* const argv) {
    if(argc < 4)
        return std::printf("Usage: %s factorize <number> <primes location>", argv[0]);

    const Aesi<512> number = std::string_view(argv[2]);
    thrust::device_vector<Aesi<512>> numberAndFactor = { number, { 0 } };
    Timer::init() << "Factorizing number " << std::hex << std::showbase << number << std::dec << '.' << Timer::endl;

    const thrust::device_vector<uint64_t> primes = loadPrimes(argv[3]);
    Timer::out << "Loaded prime table of " << primes.size() << " elements." << Timer::endl;

    kernel<<<32, 32>>>(
            thrust::raw_pointer_cast(numberAndFactor.data()),
            thrust::raw_pointer_cast(primes.data()),
            primes.size());

    const auto code = hipDeviceSynchronize();
    if (code != hipSuccess)
        return std::printf("Kernel launch failed: %s.\n", hipGetErrorString(code));
    Timer::out << "Kernel completed. Founded factor: " << std::hex << std::showbase << numberAndFactor[1] << '.' << Timer::endl;

    return 0;
}
